#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;

__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }

hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

// CUDA event ��ü�� ����Ͽ� Ŀ�� ����ð� ����
void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL(hipEventDestroy(cuda_timer_start));
	CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }


TIMER_T compute_time = 0;
TIMER_T device_time = 0;

int n;
#define BLOCK_SIZE 256
#define ELEMENT_SIZE (1<<20)
const int ELEM_PER_VECTOR = 32;
float(*pVecX), (*pVecY), (*pVecY_G);
float(*pMatA);
void init_MatVec(void)
{
	srand((unsigned)time(NULL));
	FILE* fp = fopen("gen.bin", "rb");
	fread(&n, sizeof(float), 1, fp);

	pVecX = new float[n * ELEM_PER_VECTOR];
	pVecY = new float[n * ELEM_PER_VECTOR];
	pVecY_G = new float[n * ELEM_PER_VECTOR];
	pMatA = new float[ELEM_PER_VECTOR * ELEM_PER_VECTOR];

	fread(pVecX, sizeof(float), n * ELEM_PER_VECTOR, fp);
	fread(pMatA, sizeof(float), ELEM_PER_VECTOR * ELEM_PER_VECTOR, fp);
	fclose(fp);
}

void Mat_Vec_Multiply()
{
	int vec_idx, i, j;

	for (vec_idx = 0; vec_idx < ELEMENT_SIZE; vec_idx++) {
		for (i = 0; i < ELEM_PER_VECTOR; i++) {
			float sum = 0;
			for (j = 0; j < ELEM_PER_VECTOR; j++) {
				sum += pMatA[i * ELEM_PER_VECTOR + j] * pVecX[vec_idx * ELEM_PER_VECTOR + j];
			}
			pVecY[vec_idx * ELEM_PER_VECTOR + i] = sum;
		}
	}
}

__global__ void Mat_Vec_Multiply_Kernel(float *d_VecY, float *d_VecX, float *d_MatA, int Vec_Size)
{
	//TODO
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int i, j;

	for (i = 0; i < Vec_Size; i++) {
		float sum = 0;
		for (j = 0; j < Vec_Size; j++) {
			sum += d_MatA[i * Vec_Size + j] * d_VecX[id * Vec_Size + j];
		}
		d_VecY[id * Vec_Size + i] = sum;
	}
}

void Mat_Vec_Multiply_GPU(float *p_VecX, float *p_MatA, float *p_VecY_G)
{
	float *d_VecY, *d_VecX, *d_MatA;

	//TODO
	size_t size = n * ELEM_PER_VECTOR * sizeof(float);
	CUDA_CALL(hipMalloc(&d_VecY, size));
	//CUDA_CALL(hipMemcpy(d_VecY, p_VecY_G, size, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_VecX, size));
	CUDA_CALL(hipMemcpy(d_VecX, p_VecX, size, hipMemcpyHostToDevice));

	size = ELEM_PER_VECTOR * ELEM_PER_VECTOR * sizeof(float);
	CUDA_CALL(hipMalloc(&d_MatA, size));
	CUDA_CALL(hipMemcpy(d_MatA, p_MatA, size, hipMemcpyHostToDevice));

	//�Ʒ� �ڵ带 ����Ͽ� Ŀ���� ȣ��
	
	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid(ELEMENT_SIZE / dimBlock.x);
	CHECK_TIME_INIT_GPU();
	CHECK_TIME_START_GPU();
	Mat_Vec_Multiply_Kernel << <dimGrid, dimBlock >> > (d_VecY, d_VecX, d_MatA, ELEM_PER_VECTOR);
	CHECK_TIME_END_GPU(device_time);
	CHECK_TIME_DEST_GPU();
	

	//TODO
	size = n * ELEM_PER_VECTOR * sizeof(float);
	CUDA_CALL(hipMemcpy(p_VecY_G, d_VecY, size, hipMemcpyDeviceToHost));

	hipFree(d_VecY);
	hipFree(d_VecX);
	hipFree(d_MatA);
}

void init_data(int size) {
	srand(0);
	FILE *fp = fopen("gen.bin", "wb");
	fwrite(&size, sizeof(int), 1, fp);

	int i, j;
	float x;

	for (i = 0; i < size; i++) {
		for (j = 0; j < ELEM_PER_VECTOR; j++) {
			x = 2.0f*((float)rand() / RAND_MAX) - 1.0f;
			fwrite(&x, sizeof(float), 1, fp);
		}
	}

	for (i = 0; i < ELEM_PER_VECTOR; i++) {
		for (j = 0; j < ELEM_PER_VECTOR; j++) {
			x = 2.0f*((float)rand() / RAND_MAX) - 1.0f;
			fwrite(&x, sizeof(float), 1, fp);
		}
	}

	fclose(fp);

	return;
}

bool check_equal() {
	for (int i = 0; i < ELEMENT_SIZE * ELEM_PER_VECTOR; i++) {
		if (fabs(pVecY[i] - pVecY_G[i]) > 0.001) {
			return false;
		}
	}
	return true;
}

int main()
{
	init_data(ELEMENT_SIZE);
	init_MatVec();
	printf("n = %d  file open ok.\n", n);

	CHECK_TIME_START;
	Mat_Vec_Multiply();
	CHECK_TIME_END(compute_time);
	printf("***CPU Time taken = %.6fms\n", compute_time);

	Mat_Vec_Multiply_GPU(pVecX, pMatA, pVecY_G);
	printf("***GPU Time taken = %.6fms\n", device_time);

	bool check = check_equal();
	if (check)
		printf("CPU and GPU calculate same\n");
	else
		printf("CPU and GPU calculate difference\n");

	printf("CPU [10] = %f, GPU [10] = %f\n", pVecY[10], pVecY_G[10]);

	return 0;
}


